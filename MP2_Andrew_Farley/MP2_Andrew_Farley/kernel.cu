#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// thread block size
#define BLOCKDIM 16
#define N 100

const float val1 = 4.0f;
const float val2 = 2.0f;

__global__ void MatAdd(float *a, float *b, float *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * N;
	if (i < N && j < N)
		c[index] = a[index] + b[index];
}

__global__ void MatAddRow(float *a, float *b, float *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = 0; j < N; j++) {
		int index = i + j * N;
		if (i < N && j < N)
			c[index] = a[index] + b[index];
	}
}

__global__ void MatAddCol(float *a, float *b, float *c) {
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	for (int i = 0; i < N; i++) {
		int index = i + j * N;
		if (i < N && j < N)
			c[index] = a[index] + b[index];
	}
}

typedef float myMat[N*N];

int main() {
	myMat *A, *B, *C;
	size_t dsize = N*N*sizeof(float);
	A = (myMat*)malloc(dsize);
	B = (myMat*)malloc(dsize);
	C = (myMat*)malloc(dsize);

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int index = i + j * N;
			(*A)[index] = val1;
			(*B)[index] = val2;
			(*C)[index] = 0.0f;
		}
	}

	float *pA, *pB, *pC;

	printf("cudaMalloc\n");
	// allocate matrices in device memory
	printf("First\n");
	hipMalloc((void**)&pA, (N*N)*sizeof(float));
	printf("Second\n");
	hipMalloc((void**)&pB, (N*N)*sizeof(float));
	printf("Third\n");
	hipMalloc((void**)&pC, (N*N)*sizeof(float));

	printf("cudaMemcpy\n");
	// copy matrices from host memory to device memory
	hipMemcpy(pA, A, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N)*sizeof(float), hipMemcpyHostToDevice);

	// KERNEL INVOCATION
	// each thread produces 1 output matrix element
	dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
	dim3 numBlocks((int)ceil(N / threadsPerBlock.x), (int)ceil(N / threadsPerBlock.y));
	MatAdd<<<numBlocks, threadsPerBlock>>>(pA, pB, pC);

	// copy result from device memory to host memory
	hipMemcpy(C, pC, (N*N)*sizeof(float), hipMemcpyDeviceToHost);

	int i, j;
	printf("Array C = \n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			int index = i + j * N;
			printf("%f ", C[index]);
		}
		printf("\n");
	}

	// free device memory
	hipFree(pA);
	hipFree(pB);
	hipFree(pC);

	return 0;
}
