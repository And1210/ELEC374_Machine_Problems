#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// thread block size
#define BLOCKDIM 16

// threshold
#define TOLERANCE 0.01
float absf(float n);

__global__ void MatMult(float *a, float *b, float *c, int N, int tileWidth) {
	int i = blockIdx.x * tileWidth + threadIdx.x;
	int j = blockIdx.y * tileWidth + threadIdx.y;

	int index = i + j * N;
	for (int k = 0; k < N; k++) {
		int a_index = i + k * N;
		int b_index = k + j * N;
		if (i < N && j < N) {
			c[index] += a[a_index] * b[b_index];
		}
	}
	//printf("%d %d %f\n", i, j, total);
}

typedef float myMat[];

void HostFunction(myMat* A, myMat* B, myMat* C, int N, int tileWidth);

size_t dsize;

int main() {
	myMat *A, *B, *C;

	int tileWidths[5] = { 2, 4, 10, 20, 25 };
	int Nsizes[5] = { 100, 200, 500, 1500, 5000 };

	for (int j = 0; j < 5; j++) {
		int tileWidth = tileWidths[j];
		printf("Tile Width = %d:\n", tileWidth);
		for (int i = 0; i < 4; i++) {
			int N = Nsizes[i];
			dsize = N*N*sizeof(float);
			A = (myMat*)malloc(dsize);
			B = (myMat*)malloc(dsize);
			C = (myMat*)malloc(dsize);
			printf("N = %d\n", N);
			HostFunction(A, B, C, N, tileWidth);
			printf("\n");

			free(A);
			free(B);
			free(C);
		}
		printf("\n");
	}

	//5000 matricies, they take foreverrrr
	for (int j = 0; j < 5; j++) {
		int tileWidth = tileWidths[j];
		printf("Tile Width = %d:\n", tileWidth);
		for (int i = 4; i < 5; i++) {
			int N = Nsizes[i];
			dsize = N*N*sizeof(float);
			A = (myMat*)malloc(dsize);
			B = (myMat*)malloc(dsize);
			C = (myMat*)malloc(dsize);
			printf("N = %d\n", N);
			HostFunction(A, B, C, N, tileWidth);
			printf("\n");

			free(A);
			free(B);
			free(C);
		}
		printf("\n");
	}

	getc(stdin);

	return 0;
}

void HostFunction(myMat* A, myMat* B, myMat* C, int N, int tileWidth) {
	//Initialize matricies
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int index = i + j * N;
			(*A)[index] = 10 * (float)rand() / (float)RAND_MAX;
			(*B)[index] = 10 * (float)rand() / (float)RAND_MAX;
			(*C)[index] = 0.0f;
		}
	}

	//Pointers to matricies
	float *pA, *pB, *pC;

	//Allocate matrices in device memory
	hipMalloc((void**)&pA, (N*N)*sizeof(float));
	hipMalloc((void**)&pB, (N*N)*sizeof(float));
	hipMalloc((void**)&pC, (N*N)*sizeof(float));

	/*
	float time = 0;
	cudaEvent_t start, end;
	cudaEventCreate(&start);
	cudaEventCreate(&end);
	cudaEventRecord(start);
	addHandler(pA, pB, pC, N);
	cudaEventRecord(end);
	cudaEventSynchronize(end);
	cudaEventElapsedTime(&time, start, end);
	cudaEventDestroy(start);
	cudaEventDestroy(end);
	printf("Kernal function time: %f\n", time);*/

	//Copy matrices from host memory to device memory
	float time = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	hipMemcpy(pA, A, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N)*sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	printf("Transfer to device time: %f\n", time);

	//KERNEL CALL
	//Each thread produces 1 output matrix element
	dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
	dim3 numBlocks((int)ceil(N / (float)threadsPerBlock.x), (int)ceil(N / (float)threadsPerBlock.y));
	MatMult <<<numBlocks, threadsPerBlock>>>(pA, pB, pC, N, tileWidth);

	//Copy result from device memory to host memory
	time = 0;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);
	hipMemcpy(C, pC, (N*N)*sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	printf("Transfer to host time: %f\n", time);

	//Compute matrix multiplication using the CPU
	myMat *CTemp;
	CTemp = (myMat*)malloc(dsize);
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			int index = i + j * N;
			(*CTemp)[index] = 0.0;
			for (int k = 0; k < N; k++) {
				int a_index = i + k * N;
				int b_index = k + j * N;
				(*CTemp)[index] += (*A)[a_index] * (*B)[b_index];
			}
		}
	}

	//Compare GPU computed multiplication to CPU
	int good = 1;
	int i, j;
	//printf("Array C = \n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			int index = i + j * N;
			float val = (*C)[index];
			//printf("%f ", val);
			float diff = (*CTemp)[index] - val;
			/*if (absf(diff) > TOLERANCE) {
			printf("%d %d %f %f %f\n", i, j, val, (*CTemp)[index], diff);
			good = 0;
			}*/
		}
		//printf("\n");
	}

	if (good == 1) {
		printf("TEST PASSED\n");
	}
	else {
		printf("TEST FAILED\n");
	}

	// free device memory
	hipFree(pA);
	hipFree(pB);
	hipFree(pC);
}

float absf(float n) {
	if (n < 0)
		return -n;
	return n;
}